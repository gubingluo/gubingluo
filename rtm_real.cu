#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "function.h"
#include "sgyhead.h"
#include "common.h"
#include "kernel.cuh"

// shotprofileId[5][nshot]
// 1 ��: sx             shotprofileId[0][ishot]
// 2 ��: ntrace         shotprofileId[1][ishot]
// 3 ��: itraceAll      shotprofileId[2][ishot]
// 4 ��: offset         shotprofileId[3][ishot]

extern "C" void rtm_real(int idevice, int nshot, int startshot, int dshot, int medium_flag, int flag_layout, int sgyflag, int endianflag, int ismth,
	                   int nx, int nz, int nt, float dx, float dz, float dt, int nx0, int nz0, int nt0, float dx0, float dz0, float dt0,
		             int npml, int lx, int lz, float tsnap, float fdom, float spz0, float amp, float alp, float direct, float direct0,  
		             float *vp0, float *rho0, float *diffcoef, float *wavelet, char filename_shotgather[40], int **shotprofileId, 
				 float *image1,  float *image2s, float *image3s, float *image4s, float *image5s, 
		 		 float *image2m, float *image3m, float *image4m, float *image5m, float *illum)
{
// common parameter
	int nxpad,nzpad;
	int ntsnap = (int)(tsnap/dt);
	int nw = (int)(direct/(fdom*dt)),
	    tlength = (int)(direct0/dt);
	int it,ix,ishot;
	long long offset_trace;
	FILE *fp;	
// wavefield extrapolation parameter
	float mstimer;
	char buffrecord[40];
	float vpmax;
	float spx,spz;

	static float dx2,dz2,_dt,_dtx,_dtz;
	static int nsx,nsz,nxz;

	static dim3 dimGrid,dimBlock,
		      dimGridp,dimGridvx,dimGridvz,dimGridvxb,dimGridvzb,
		      dimGridvxlr,dimGridvztb,
		      dimGridplr,dimGridptb,
			dimGridpcooner,
		      dimGridrecord,
			dimGridpmllr,dimGridpmltb,dimBlockpmllr,dimBlockpmltb,
			dimGridfull;

	// variables on host
	float *vp,*rho,*temp,*record,*tr;
	float *vxspmllr,*vzspmltb,*pspmllr,*pspmltb;
	float *p;

	// variables on device
	float *d_wavelet,*d_diffcoef;
	float *d_source,*d_record,*d_vp,*d_rho;
	float *d_p,*d_vx,*d_vz,*d_p_pre,
		*d_ps,*d_vxs,*d_vzs,*d_ps_pre;
	float *d_pl1,*d_pl2,*d_pr1,*d_pr2,
	      *d_pt1,*d_pt2,*d_pb1,*d_pb2;
	float *d_ddx,*d_ddz,*d_ddxVx,*d_ddzVz;
	int *d_norder,*d_norderx,*d_norderz;

	float *d_vxspmllr,*d_vzspmltb,*d_pspmllr,*d_pspmltb;
	// single shot image
	float *d_g1,                                 // cross-coorelation
		*d_g2,*d_g2ud,*d_g2du,*d_g2lr,*d_g2rl, // wavefiled-decomposition
		*d_g3,*d_g3_true,*d_g31,*d_g32,                   // poynting vector   d_g32 true amplitude
		*d_g4;                                 // energy norm

	float *d_image1,
		*d_image2s,*d_image3s,*d_image4s,*d_image5s,
		*d_image2m,*d_image3m,*d_image4m,*d_image5m,
		*d_imagetrue,*d_illum; 
	float *imagetrue,*d_Illum;        
	float *d_taper;
	int nwin=50;
	float alpha=0.06;

	float **seisobs,**seiscal;

      _dtx = dt/dx;
	_dtz = dt/dz;
	_dt  = 1.0/dt;
	dx2 = dx*dx;
	dz2 = dz*dz;

	hipSetDevice(idevice);
	check_gpu_error("Failed to initialize device");
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&d_wavelet,       nt*sizeof(float));
	hipMemset(d_wavelet,   0,   nt*sizeof(float));
	cuda_ricker_wavelet<<<(nt+511)/512,512>>>(d_wavelet, fdom, dt, nt);

	hipMalloc(&d_image1,       nx*nz*sizeof(float));
	hipMalloc(&d_image2s,      nx*nz*sizeof(float));	
	hipMalloc(&d_image3s,      nx*nz*sizeof(float));	
	hipMalloc(&d_image4s,      nx*nz*sizeof(float));	
	hipMalloc(&d_image5s,      nx*nz*sizeof(float));	
	hipMalloc(&d_image2m,      nx*nz*sizeof(float));	
	hipMalloc(&d_image3m,      nx*nz*sizeof(float));	
	hipMalloc(&d_image4m,      nx*nz*sizeof(float));	
	hipMalloc(&d_image5m,      nx*nz*sizeof(float));	
	hipMalloc(&d_Illum,        nx*nz*sizeof(float));
	hipMalloc(&d_imagetrue,    nx*nz*sizeof(float));	
	hipMemset(d_image1,    0,  nx*nz*sizeof(float));
	hipMemset(d_image2s,   0,  nx*nz*sizeof(float));
	hipMemset(d_image3s,   0,  nx*nz*sizeof(float));
	hipMemset(d_image4s,   0,  nx*nz*sizeof(float));
	hipMemset(d_image5s,   0,  nx*nz*sizeof(float));
	hipMemset(d_image2m,   0,  nx*nz*sizeof(float));
	hipMemset(d_image3m,   0,  nx*nz*sizeof(float));
	hipMemset(d_image4m,   0,  nx*nz*sizeof(float));
	hipMemset(d_image5m,   0,  nx*nz*sizeof(float));
	hipMemset(d_Illum,     0,  nx*nz*sizeof(float));
	hipMemset(d_imagetrue, 0,  nx*nz*sizeof(float));

	imagetrue=(float *)malloc(nx*nz*sizeof(float));
	memset(imagetrue, 0, nx*nz*sizeof(float));
	for (ishot = startshot; ishot <= nshot; ishot=ishot+dshot)
	{
		hipEventRecord(start);
		//==============================================================================
		//�������У�����λ���Ҳ࣬��Сƫ�ƾ�Ϊoffset��
		int noffset,nx1,nx2,nxlength;
		int noffset0,nx10,nx20,nxlength0;
		float offsetmax,offset;

		offsetmax = (float)(shotprofileId[3][ishot-1] + (shotprofileId[1][ishot-1] - 1)*dx0);
		offset    = (float)(shotprofileId[3][ishot-1]);
		spx       = (float)(shotprofileId[0][ishot-1]);
		spz       = spz0;
		// given
		noffset0 = (int)(offset/dx0);
		nx10 = (int)(spx/dx0);
		nx20 = (int)((spx + offsetmax)/dx0);
		nxlength0 = nx20 - nx10 + 1;
		// optimal
		noffset = (int)(offset/dx);
		nx1 = (int)(spx/dx);
		nx2 = (int)((spx + offsetmax)/dx);
		nxlength = nx2 - nx1 + 1;
		// optimal extend
		spx = spx - nx1*dx;   // local
		spx = spx + npml*dx;
		spz = spz + npml*dz;
		nxpad = nxlength + 2*npml;
		nzpad = nz + 2*npml;
			
		nsx = (int)(spx/dx);				
		nsz = (int)(spz/dz);
		nxz = nxpad*nzpad;
		// block and thread
		dimBlock = dim3(Block_Sizez, Block_Sizex);
		dimBlockpmllr = dim3(Block_Sizez,N/2);
		dimBlockpmltb = dim3(N/2,Block_Sizex);
		dimGrid  = dim3((nzpad+Block_Sizez-1)/Block_Sizez,           (nxpad+Block_Sizex-1)/Block_Sizex); 
            dimGridp  = dim3((nzpad-2*npml+Block_Sizez-1)/Block_Sizez,   (nxpad-2*npml+Block_Sizex-1)/Block_Sizex);
		dimGridvx = dim3((nzpad+Block_Sizez-1)/Block_Sizez,          (nxpad-2*npml-1+Block_Sizex-1)/Block_Sizex);
		dimGridvz = dim3((nzpad-2*npml-1+Block_Sizez-1)/Block_Sizez, (nxpad+Block_Sizex-1)/Block_Sizex);
		dimGridvxb = dim3((nzpad-2*npml+Block_Sizez-1)/Block_Sizez,  (nxpad-2*npml-1+Block_Sizex-1)/Block_Sizex);
		dimGridvzb = dim3((nzpad-2*npml-1+Block_Sizez-1)/Block_Sizez,(nxpad-2*npml+Block_Sizex-1)/Block_Sizex);

		dimGridvxlr = dim3((nzpad+Block_Sizez-1)/Block_Sizez,2);
		dimGridvztb = dim3(2,(nxpad+Block_Sizex-1)/Block_Sizex);
		dimGridplr  = dim3((nzpad-2*npml+Block_Sizez-1)/Block_Sizez,2);
		dimGridptb  = dim3(2,(nxpad-2*npml+Block_Sizex-1)/Block_Sizex);
		dimGridpcooner = dim3(2,2);

		dimGridrecord = dim3((nt+Block_Sizez-1)/Block_Sizez,     (nxpad-2*npml-noffset+Block_Sizex-1)/Block_Sizex);
		dimGridpmllr  = dim3((nzpad-2*npml+Block_Sizez-1)/Block_Sizez,2);
		dimGridpmltb  = dim3(2,(nxpad-2*npml+Block_Sizex-1)/Block_Sizex);
		//ģ�����������׼��
		record = (float *)malloc(nt*(nxlength-noffset)*sizeof(float));
		temp = (float *)malloc(nz*nxlength*sizeof(float));
		vp = (float *)malloc(nzpad*nxpad*sizeof(float));
		rho= (float *)malloc(nzpad*nxpad*sizeof(float));			
		p  = (float *)malloc(nzpad*nxpad*sizeof(float));
		vxspmllr = (float *)malloc(N*(nzpad-2*npml)*nt*sizeof(float));
		vzspmltb = (float *)malloc(N*(nxpad-2*npml)*nt*sizeof(float));
		pspmllr = (float *)malloc(N*(nzpad-2*npml)*nt*sizeof(float));
		pspmltb = (float *)malloc(N*(nxpad-2*npml)*nt*sizeof(float));
		tr = (float *)malloc(nt0*sizeof(float));
						
		memset(record, 0, nt*(nxlength-noffset)*sizeof(float));
		memset(temp,   0, nz*nxlength*sizeof(float));
		memset(vp,     0, nxpad*nzpad*sizeof(float));
		memset(rho,    0, nxpad*nzpad*sizeof(float));
		memset(p,      0, nxpad*nzpad*sizeof(float));
		memset(vxspmllr,0, N*(nzpad-2*npml)*nt*sizeof(float));
		memset(vzspmltb,0, N*(nxpad-2*npml)*nt*sizeof(float));
		memset(pspmllr, 0, N*(nzpad-2*npml)*nt*sizeof(float));
		memset(pspmltb, 0, N*(nxpad-2*npml)*nt*sizeof(float));
		//===============================================================================
		extractvel1(temp, vp0,  nx, nz, nx1, nx2);
		extendvel1(vp,  temp, nxlength, nz, npml);	
		extractrho1(temp, rho0, nx, nz, nx1, nx2);					
		extendvel1(rho, temp, nxlength, nz, npml);
		free(temp);
		// pml layers smooth
		if (medium_flag){
			pmlvelsmooth1d(vp,  nxpad, nzpad, npml);
				pmlvelsmooth1d(rho, nxpad, nzpad, npml);}		
		vpmax = sqrtf(Maxval1(vp, nzpad*nxpad));

		// alloc device memory
		hipMalloc(&d_diffcoef,   (N/2)*(N/2)*sizeof(float));
		hipMalloc(&d_record,     (nxlength-noffset)*nt*sizeof(float));
		hipMalloc(&d_source,     nxz*sizeof(float));
		hipMalloc(&d_vp,         nxz*sizeof(float));
		hipMalloc(&d_rho,        nxz*sizeof(float));

		hipMalloc(&d_p,          nxz*sizeof(float));
		hipMalloc(&d_p_pre,      nxz*sizeof(float));
		hipMalloc(&d_vx,         (nxpad-1)*nzpad*sizeof(float));
		hipMalloc(&d_vz,         nxpad*(nzpad-1)*sizeof(float));
		hipMalloc(&d_ps,         nxz*sizeof(float));
		hipMalloc(&d_ps_pre,     nxz*sizeof(float));
		hipMalloc(&d_vxs,        (nxpad-1)*nzpad*sizeof(float));
		hipMalloc(&d_vzs,        nxpad*(nzpad-1)*sizeof(float));
		
		hipMalloc(&d_pl1,        npml*nzpad*sizeof(float));
		hipMalloc(&d_pl2,        npml*nzpad*sizeof(float));
		hipMalloc(&d_pr1,        npml*nzpad*sizeof(float));
		hipMalloc(&d_pr2,        npml*nzpad*sizeof(float));
		hipMalloc(&d_pt1,        npml*(nxpad-2*npml)*sizeof(float));
		hipMalloc(&d_pt2,        npml*(nxpad-2*npml)*sizeof(float));
		hipMalloc(&d_pb1,        npml*(nxpad-2*npml)*sizeof(float));
		hipMalloc(&d_pb2,        npml*(nxpad-2*npml)*sizeof(float));

		hipMalloc(&d_ddx,        nxpad*sizeof(float));
		hipMalloc(&d_ddz,        nzpad*sizeof(float));
		hipMalloc(&d_ddxVx,      (nxpad-1)*sizeof(float));
		hipMalloc(&d_ddzVz,      (nzpad-1)*sizeof(float));
		hipMalloc(&d_norder,     nxz*sizeof(int));
		hipMalloc(&d_norderx,    (nxpad-1)*sizeof(int));
		hipMalloc(&d_norderz,    (nzpad-1)*sizeof(int));

		hipMalloc(&d_vxspmllr,   N*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_vzspmltb,   N*(nxpad-2*npml)*sizeof(float));
		hipMalloc(&d_pspmllr,    N*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_pspmltb,    N*(nxpad-2*npml)*sizeof(float));	
			
		hipMalloc(&d_g1,          (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMalloc(&d_g2,          (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));		
		hipMalloc(&d_g3,          (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMalloc(&d_g3_true,     (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMalloc(&d_g4,          (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_g2ud,        (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMalloc(&d_g2du,        (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_g2lr,        (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_g2rl,        (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_g31,         (nxpad-2*npml)*(nzpad-2*npml)*61*sizeof(float));	
		hipMalloc(&d_g32,         (nxpad-2*npml)*(nzpad-2*npml)*61*sizeof(float));
		hipMalloc(&d_illum,       (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMalloc(&d_taper,       (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	

		hipMemcpy(d_diffcoef, diffcoef, (N/2)*(N/2)*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_vp,       vp,       nxz*sizeof(float),         hipMemcpyHostToDevice);
		hipMemcpy(d_rho,      rho,      nxz*sizeof(float),         hipMemcpyHostToDevice);
		
		hipMemset(d_record, 0, (nxlength-noffset)*nt*sizeof(float));
		hipMemset(d_p,      0, nxpad*nzpad*sizeof(float));
		hipMemset(d_vx,     0, (nxpad-1)*nzpad*sizeof(float));
		hipMemset(d_vz,     0, nxpad*(nzpad-1)*sizeof(float));
		hipMemset(d_p_pre,  0, nxpad*nzpad*sizeof(float));
		hipMemset(d_ps,     0, nxpad*nzpad*sizeof(float));
		hipMemset(d_vxs,    0, (nxpad-1)*nzpad*sizeof(float));
		hipMemset(d_vzs,    0, nxpad*(nzpad-1)*sizeof(float));
		hipMemset(d_ps_pre, 0, nxpad*nzpad*sizeof(float));		

		hipMemset(d_pl1,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pl2,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pr1,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pr2,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pt1,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pt2,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pb1,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pb2,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_ddx,    0, nxpad*sizeof(float));
		hipMemset(d_ddz,    0, nzpad*sizeof(float));
		hipMemset(d_ddxVx,  0, (nxpad-1)*sizeof(float));
		hipMemset(d_ddzVz,  0, (nzpad-1)*sizeof(float));
		hipMemset(d_norder, 0, nxpad*nzpad*sizeof(int));
		hipMemset(d_norderx,0, (nxpad-1)*sizeof(int));
		hipMemset(d_norderz,0, (nzpad-1)*sizeof(int));
		hipMemset(d_vxspmllr,0, N*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_vzspmltb,0, N*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pspmllr, 0, N*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_pspmltb, 0, N*(nxpad-2*npml)*sizeof(float));

		hipMemset(d_g1,      0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMemset(d_g2,      0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));	
		hipMemset(d_g3,      0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g3_true, 0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g4,      0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g2ud,    0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g2du,    0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g2lr,    0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g2rl,    0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_g31,     0, (nxpad-2*npml)*(nzpad-2*npml)*61*sizeof(float));
		hipMemset(d_g32,     0, (nxpad-2*npml)*(nzpad-2*npml)*61*sizeof(float));
		hipMemset(d_illum,   0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_taper,   0, (nxpad-2*npml)*(nzpad-2*npml)*sizeof(float));

		cuda_source<<<dimGrid,dimBlock>>>(d_source, nsx, nsz, nxpad, nzpad, amp, alp, dx2, dz2);
		cuda_pmlCoeffpx<<<(nxpad+127)/128,128>>>(d_ddx, vpmax, dx, npml, nxpad);
		cuda_pmlCoeffpz<<<(nzpad+127)/128,128>>>(d_ddz, vpmax, dz, npml, nzpad);
		cuda_pmlCoeffvx<<<(nxpad+126)/128,128>>>(d_ddxVx, vpmax, dx, npml, nxpad-1);
		cuda_pmlCoeffvz<<<(nzpad+126)/128,128>>>(d_ddzVz, vpmax, dz, npml, nzpad-1);
		cuda_norder<<<dimGrid,dimBlock>>>       (d_norder, nxpad, nzpad);
		cuda_norderx<<<(nxpad+126)/128,128>>>   (d_norderx, nxpad-1);
		cuda_norderz<<<(nzpad+126)/128,128>>>   (d_norderz, nzpad-1);	
		while (2*nwin >= nxlength)
			nwin -= 3;
		cuda_taper_calculate<<<dimGridp,dimBlock>>>(d_taper, nxpad-2*npml, nzpad-2*npml, nwin, alpha);
		printf("N0. %d shot source wavefield calculating......\n",ishot);			
		// calculate source wavefiled to save pml layer
		for (it=0; it<nt; it++)
		{
			if (it%ntsnap == 0){
				hipMemcpy(p, d_p, nxz*sizeof(float), hipMemcpyDeviceToHost);
				printf("source-the current shot: %d\ttime: %f s; wavefield: %.5e\n",ishot,it*dt, absMaxval1(p, nxz));}
			cuda_forward_vx<<<dimGridvx,dimBlock>>>(d_p, d_vx, d_rho, d_diffcoef, _dtx, npml, nxpad, nzpad);
			cuda_forward_vz<<<dimGridvz,dimBlock>>>(d_p, d_vz, d_rho, d_diffcoef, _dtz, npml, nxpad, nzpad);
			cuda_pml_vxlr<<<dimGridvxlr,dimBlock>>>(d_p, d_vx, d_rho, d_diffcoef, d_ddxVx, _dtx, dt, npml, nxpad, nzpad, d_norderx);
			cuda_pml_vztb<<<dimGridvztb,dimBlock>>>(d_p, d_vz, d_rho, d_diffcoef, d_ddzVz, _dtz, dt, npml, nxpad, nzpad, d_norderz);
			cuda_forward_p<<<dimGridp,dimBlock>>>(d_p, d_vx, d_vz, d_rho, d_vp, d_diffcoef, _dtx, _dtz, npml, nxpad, nzpad);
			cuda_pml_plr<<<dimGridplr,dimBlock>>>(d_p, d_vx, d_vz, d_pl1, d_pl2, d_pr1, d_pr2, d_rho, d_vp, d_diffcoef, d_ddx, d_ddz,_dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			cuda_pml_ptb<<<dimGridptb,dimBlock>>>(d_p, d_vx, d_vz, d_pt1, d_pt2, d_pb1, d_pb2, d_rho, d_vp, d_diffcoef, d_ddz, _dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			cuda_pml_pconner<<<dimGridpcooner,dimBlock>>>(d_p, d_vx, d_vz, d_pl1, d_pl2, d_pr1, d_pr2, d_rho, d_vp, d_diffcoef, d_ddx, d_ddz,_dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			cuda_add_source<<<dimGrid,dimBlock>>>(d_p, d_source, d_wavelet, dt, 1, nxpad, nzpad, it);

			save_d_vxpml<<<dimGridpmllr,dimBlockpmllr>>>(d_vx, d_vxspmllr, nxpad, nzpad, npml);
			save_d_vzpml<<<dimGridpmltb,dimBlockpmltb>>>(d_vz, d_vzspmltb, nxpad, nzpad, npml);
			save_d_ppmllr<<<dimGridpmllr,dimBlockpmllr>>>(d_p, d_pspmllr,  nxpad, nzpad, npml);
			save_d_ppmltb<<<dimGridpmltb,dimBlockpmltb>>>(d_p, d_pspmltb,  nxpad, nzpad, npml);
			hipMemcpy(&vxspmllr[it*N*(nzpad-2*npml)],       d_vxspmllr, N*(nzpad-2*npml)*sizeof(float),       hipMemcpyDeviceToHost);
			hipMemcpy(&vzspmltb[it*N*(nxpad-2*npml)],       d_vzspmltb, N*(nxpad-2*npml)*sizeof(float),       hipMemcpyDeviceToHost);	
			hipMemcpy(&pspmllr[it*N*(nzpad-2*npml)],        d_pspmllr,  N*(nzpad-2*npml)*sizeof(float),       hipMemcpyDeviceToHost);
			hipMemcpy(&pspmltb[it*N*(nxpad-2*npml)],        d_pspmltb,  N*(nxpad-2*npml)*sizeof(float),       hipMemcpyDeviceToHost);
		}

		// initial source wavefiled
		// save last snap used to reconstruction source wavefield
		hipMemcpy(d_ps, d_p,  nxz*sizeof(float), hipMemcpyDeviceToDevice);
		// initial receiver wavefield
		hipMemset(d_p,      0, nxpad*nzpad*sizeof(float));
		hipMemset(d_vx,     0, (nxpad-1)*nzpad*sizeof(float));
		hipMemset(d_vz,     0, nxpad*(nzpad-1)*sizeof(float));
		hipMemset(d_pl1,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pl2,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pr1,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pr2,    0, npml*nzpad*sizeof(float));
		hipMemset(d_pt1,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pt2,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pb1,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pb2,    0, npml*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_vxspmllr,0, N*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_vzspmltb,0, N*(nxpad-2*npml)*sizeof(float));
		hipMemset(d_pspmllr, 0, N*(nzpad-2*npml)*sizeof(float));
		hipMemset(d_pspmltb, 0, N*(nxpad-2*npml)*sizeof(float));		


		if (ismth)
		{
			velsmooth1d(vp,  nzpad, nxpad, ismth);
			velsmooth1d(rho, nzpad, nxpad, ismth);
			hipMemcpy(d_vp,       vp,       nxz*sizeof(float),         hipMemcpyHostToDevice);
			hipMemcpy(d_rho,      rho,      nxz*sizeof(float),         hipMemcpyHostToDevice);
		}
		//===============================================================================
		// prepare seismic profile
		seisobs=Creat2dArray(nt0,nxlength0-noffset0);
		seiscal=Creat2dArray(nt, nxlength-noffset);
		sprintf(buffrecord,"./seisReal/%s",filename_shotgather);		
		offset_trace = (shotprofileId[2][ishot-1] - 1)*(240 + nt0*sizeof(float)) + 3600*sgyflag;
		fp = fopen(buffrecord,"rb");
		fseek(fp,offset_trace,0);
		for (ix=0; ix<shotprofileId[1][ishot-1]; ix++)
		{		
			fseek(fp,240L,1);
			fread(tr,sizeof(float),nt0,fp);
			float_to_float(tr, nt0, endianflag);
			for (it=0; it<nt0; it++)
				seisobs[it][ix] = tr[it];

		}
		fclose(fp);
		Interpseis2d(seiscal,seisobs,nxlength-noffset,nxlength0-noffset0,nt,nt0,dt,dt0);
		for (it=0;it<nt;it++)
			for (ix=0;ix<nxlength-noffset;ix++)
				record[it*(nxlength-noffset)+ix] = seiscal[it][ix];

		free2dArray(seiscal, nt,  nxlength-noffset);
		free2dArray(seisobs, nt0, nxlength0-noffset0);
		hipMemcpy(d_record,  record,    (nxlength-noffset)*nt*sizeof(float), hipMemcpyHostToDevice);
		cuda_mute2<<<dimGridrecord,dimBlock>>>(d_record, d_vp, nsx, nsz, nt, npml, nxlength, noffset, nw, tlength, fdom, dx2, dz2, _dt);		
		// implement RTM				
		// insert seismic record for the last time slice
		cuda_insert_record2<<<(nxlength-noffset+127)/128,128>>>(d_p, &d_record[(nt-1)*(nxlength - noffset)], npml, nxlength, noffset, dt);
		// backforward record
		printf("source wavefiled prepared over...\nBegin backward......\n");
		for (it=nt-2; it>=0; it--)
		{
			if (it%ntsnap == 0){
				hipMemcpy(p, d_p, nxz*sizeof(float), hipMemcpyDeviceToHost);
				printf("backward-the current shot: %d\ttime: %f s; wavefield: %.5e\n",ishot, it*dt, absMaxval1(p, nxz));}	

			hipMemcpy(d_ps_pre, d_ps, nxz*sizeof(float), hipMemcpyDeviceToDevice);	
			hipMemcpy(d_p_pre,  d_p,  nxz*sizeof(float), hipMemcpyDeviceToDevice);
			
			// source wavefield 1: read vx vz pml; 2: calculate inner vx vz; 3: read p pml; 4; calculate inner p
			// 1  --  2
			hipMemcpy(d_vxspmllr, &vxspmllr[(it+1)*N*(nzpad-2*npml)],       N*(nzpad-2*npml)*sizeof(float),       hipMemcpyHostToDevice);
			hipMemcpy(d_vzspmltb, &vzspmltb[(it+1)*N*(nxpad-2*npml)],       N*(nxpad-2*npml)*sizeof(float),       hipMemcpyHostToDevice);				
			read_d_vxpml<<<dimGridpmllr,dimBlockpmllr>>>(d_vxs, d_vxspmllr, nxpad, nzpad, npml);
			read_d_vzpml<<<dimGridpmltb,dimBlockpmltb>>>(d_vzs, d_vzspmltb, nxpad, nzpad, npml);					
			cuda_backward_vx<<<dimGridvxb,dimBlock>>>(d_ps, d_vxs, d_rho, d_diffcoef, _dtx, npml, nxpad, nzpad);
			cuda_backward_vz<<<dimGridvzb,dimBlock>>>(d_ps, d_vzs, d_rho, d_diffcoef, _dtz, npml, nxpad, nzpad);
			// 3  --  4
			hipMemcpy(d_pspmllr, &pspmllr[it*N*(nzpad-2*npml)],        N*(nzpad-2*npml)*sizeof(float),       hipMemcpyHostToDevice);
			hipMemcpy(d_pspmltb, &pspmltb[it*N*(nxpad-2*npml)],        N*(nxpad-2*npml)*sizeof(float),       hipMemcpyHostToDevice);
			read_d_ppmllr<<<dimGridpmllr,dimBlockpmllr>>>(d_ps, d_pspmllr, nxpad, nzpad, npml);
			read_d_ppmltb<<<dimGridpmltb,dimBlockpmltb>>>(d_ps, d_pspmltb, nxpad, nzpad, npml);	
			cuda_backward_p<<<dimGridp,dimBlock>>>(d_ps, d_vxs, d_vzs, d_rho, d_vp, d_diffcoef, _dtx, _dtz, npml, nxpad, nzpad);
			// insert source 
			cuda_add_source<<<dimGrid,dimBlock>>>(d_ps, d_source, d_wavelet, dt, 2, nxpad, nzpad, it);
			// receiver wavefield
			cuda_forward_vx<<<dimGridvx,dimBlock>>>(d_p, d_vx, d_rho, d_diffcoef, _dtx, npml, nxpad, nzpad);
			cuda_forward_vz<<<dimGridvz,dimBlock>>>(d_p, d_vz, d_rho, d_diffcoef, _dtz, npml, nxpad, nzpad);
			cuda_pml_vxlr<<<dimGridvxlr,dimBlock>>>(d_p, d_vx, d_rho, d_diffcoef, d_ddxVx, _dtx, dt, npml, nxpad, nzpad, d_norderx);
			cuda_pml_vztb<<<dimGridvztb,dimBlock>>>(d_p, d_vz, d_rho, d_diffcoef, d_ddzVz, _dtz, dt, npml, nxpad, nzpad, d_norderz);

			cuda_forward_p<<<dimGridp,dimBlock>>>(d_p, d_vx, d_vz, d_rho, d_vp, d_diffcoef, _dtx, _dtz, npml, nxpad, nzpad);
			cuda_pml_plr<<<dimGridplr,dimBlock>>>  (d_p, d_vx, d_vz, d_pl1, d_pl2, d_pr1, d_pr2, d_rho, d_vp, d_diffcoef, d_ddx, d_ddz,_dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			cuda_pml_ptb<<<dimGridptb,dimBlock>>>  (d_p, d_vx, d_vz, d_pt1, d_pt2, d_pb1, d_pb2, d_rho, d_vp, d_diffcoef, d_ddz, _dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			cuda_pml_pconner<<<dimGridpcooner,dimBlock>>>(d_p, d_vx, d_vz, d_pl1, d_pl2, d_pr1, d_pr2, d_rho, d_vp, d_diffcoef, d_ddx, d_ddz,_dtx, _dtz, dt, npml, nxpad, nzpad, d_norder);
			// insert source 
			cuda_insert_record2<<<(nxlength-noffset+127)/128,128>>>(d_p, &d_record[it*(nxlength-noffset)], npml, nxlength, noffset, dt);
			// imaging condition:
			cuda_cross_coorelation<<<dimGridp,dimBlock>>>(d_ps,d_p,d_g1,d_illum,nxpad,nzpad,npml);
			cuda_wavefield_decomposition<<<dimGridp,dimBlock>>>(d_ps,d_vxs,d_vzs,d_p,d_vx,d_vz,d_g2ud,d_g2du,d_g2lr,d_g2rl,nxpad,nzpad,npml);
			cuda_poynting<<<dimGridp,dimBlock>>>(d_ps,d_vxs,d_vzs,d_p,d_vx,d_vz,d_vp,d_g31,d_g32,nxpad,nzpad,npml);
			cuda_energynorm<<<dimGridp,dimBlock>>>(d_ps,d_ps_pre,d_p,d_p_pre,d_vp,d_g4,dx,dz,dt,nxpad,nzpad,npml);
		}
		// abtain g2 and g3
		cuda_stack_udlr<<<dimGridp,dimBlock>>>(d_g2ud,d_g2du,d_g2lr,d_g2rl,d_g2,nxlength,nz);
		cuda_stack_theta<<<dimGridp,dimBlock>>>(d_g31,d_g3,nxlength,nz);
		cuda_stack_theta<<<dimGridp,dimBlock>>>(d_g32,d_g3_true,nxlength,nz);
		// taper image
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_g1, d_taper, nxlength, nz);
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_g2, d_taper, nxlength, nz);
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_g3, d_taper, nxlength, nz);
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_g4, d_taper, nxlength, nz);
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_g3_true, d_taper, nxlength, nz);
		cuda_taper_image<<<dimGridp,dimBlock>>>(d_illum, d_taper, nxlength, nz);
		// single-shot normalized			
		cuda_applyics<<<dimGridp,dimBlock>>>(d_image2s,d_g1,d_illum,nx,nxlength,nx1,nz);
		cuda_applyics<<<dimGridp,dimBlock>>>(d_image3s,d_g2,d_illum,nx,nxlength,nx1,nz);
		cuda_applyics<<<dimGridp,dimBlock>>>(d_image4s,d_g3,d_illum,nx,nxlength,nx1,nz);	
		cuda_applyics<<<dimGridp,dimBlock>>>(d_image5s,d_g4,d_illum,nx,nxlength,nx1,nz);	
		// multi-shot normalized
		cuda_applyic<<<dimGridp,dimBlock>>> (d_image1,  d_g1,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_image2m, d_g1,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_image3m, d_g2,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_image4m, d_g3,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_image5m, d_g4,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_imagetrue, d_g3_true,nx,nxlength,nx1,nz);
		cuda_applyic<<<dimGridp,dimBlock>>> (d_Illum,   d_illum,nx,nxlength,nx1,nz);
		// output temp image
		if ((ishot-1)%50 == 0)
		{		
			// single-shot normalized	
			// Image2s
			hipMemcpy(image2s, d_image2s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image2stemp.su");
			Output1d(image2s, nz, nx, dx, buffrecord, 1);
			// Image3s
			hipMemcpy(image3s, d_image3s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image3stemp.su");
			Output1d(image3s, nz, nx, dx, buffrecord, 1);
			// Image4s
			hipMemcpy(image4s, d_image4s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image4stemp.su");
			Output1d(image4s, nz, nx, dx, buffrecord, 1);
			// Image5s
			hipMemcpy(image5s, d_image5s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image5stemp.su");
			Output1d(image5s, nz, nx, dx, buffrecord, 1);		
			// multishot normlized
			// Image1
			hipMemcpy(image1, d_image1, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image1temp.su");
			Output1d(image1, nz, nx, dx, buffrecord, 1);
			// Illum
			hipMemcpy(illum,  d_Illum,  nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Illumtemp.su");
			Output1d(illum, nz, nx, dx, buffrecord, 1);
			// Image2m
			hipMemcpy(image2m, d_image2m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image2mtemp.su");
			Output1d(image2m, nz, nx, dx, buffrecord, 1);
			// Image3m
			hipMemcpy(image3m, d_image3m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image3mtemp.su");
			Output1d(image3m, nz, nx, dx, buffrecord, 1);
			// Image4m
			hipMemcpy(image4m, d_image4m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image4mtemp.su");
			Output1d(image4m, nz, nx, dx, buffrecord, 1);
			// Image5m
			hipMemcpy(image5m, d_image5m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Image5mtemp.su");
			Output1d(image5m, nz, nx, dx, buffrecord, 1);
			// Imagetrue
			hipMemcpy(imagetrue, d_imagetrue, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
			sprintf(buffrecord,"./output/Imagetruetemp.su");
			Output1d(imagetrue, nz, nx, dx, buffrecord, 1);	
		}
		free(record);
		free(vp);
		free(rho);   		   
		free(vxspmllr);
		free(vzspmltb);
		free(pspmllr);
		free(pspmltb);
		free(p);
		free(tr);

		hipFree(d_diffcoef);hipFree(d_record);hipFree(d_source);
		hipFree(d_vp);hipFree(d_rho);
		hipFree(d_p); hipFree(d_vx); hipFree(d_vz); hipFree(d_p_pre);
		hipFree(d_ps);hipFree(d_vxs);hipFree(d_vzs);hipFree(d_ps_pre);
		hipFree(d_pl1);hipFree(d_pl2);hipFree(d_pr1);hipFree(d_pr2);
		hipFree(d_pt1);hipFree(d_pt2);hipFree(d_pb1);hipFree(d_pb2);

		hipFree(d_ddx);hipFree(d_ddz);hipFree(d_ddxVx);hipFree(d_ddzVz);
		hipFree(d_norder);hipFree(d_norderx);hipFree(d_norderz);

		hipFree(d_vxspmllr);hipFree(d_vzspmltb);hipFree(d_pspmllr);hipFree(d_pspmltb);	

		hipFree(d_g1);  hipFree(d_g2);  hipFree(d_g3);  hipFree(d_g4);	
		hipFree(d_g2ud);hipFree(d_g2du);hipFree(d_g2lr);hipFree(d_g2rl);	
		hipFree(d_g31); hipFree(d_g32); hipFree(d_g3_true);		
		hipFree(d_illum);
		hipFree(d_taper);

		hipEventRecord(stop); 
		hipEventSynchronize(stop); 
	   	hipEventElapsedTime(&mstimer, start, stop); 

		printf("%d shot finished: %g (s)\n",ishot, mstimer*1.e-3); 
	}
	hipMemcpy(imagetrue, d_imagetrue, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	sprintf(buffrecord,"./output/%dImagetrue.su",idevice);
	Output1d(imagetrue, nz, nx, dx, buffrecord, 1);	
	
	hipMemcpy(image2s, d_image2s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image3s, d_image3s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image4s, d_image4s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image5s, d_image5s, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(illum,   d_Illum,   nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image1,  d_image1,  nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image2m, d_image2m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image3m, d_image3m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image4m, d_image4m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image5m, d_image5m, nx*nz*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_image1);
	hipFree(d_image2s);hipFree(d_image3s);hipFree(d_image4s);hipFree(d_image5s);
	hipFree(d_image2m);hipFree(d_image3m);hipFree(d_image4m);hipFree(d_image5m);
	hipFree(d_Illum);
	hipFree(d_imagetrue);
	free(imagetrue);
}
